#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_common.cuh"

// Template kernel for both float and double computations
template<typename T>
__global__ void lots_of_compute(T *inputs, int N, size_t niters, T *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    for (; tid < N; tid += nthreads)
    {
        size_t iter;
        T val = inputs[tid];

        for (iter = 0; iter < niters; iter++)
        {
            val = (val + T(5.0)) - T(101.0);
            val = (val / T(3.0)) + T(102.0);
            val = (val + T(1.07)) - T(103.0);
            val = (val / T(1.037)) + T(104.0);
            val = (val + T(3.00)) - T(105.0);
            val = (val / T(0.22)) + T(106.0);
        }

        outputs[tid] = val;
    }
}

int main(int argc, char **argv)
{
    double meanFloatToDeviceTime, meanFloatKernelTime, meanFloatFromDeviceTime;
    double meanDoubleToDeviceTime, meanDoubleKernelTime,
        meanDoubleFromDeviceTime;
    struct hipDeviceProp_t deviceProperties;
    size_t totalMem, freeMem;
    float *floatSample;
    double *doubleSample;
    int sampleLength = 10;
    int nRuns = 5;
    int nKernelIters = 20;

    gpuErrchk(hipMemGetInfo(&freeMem, &totalMem));
    gpuErrchk(hipGetDeviceProperties(&deviceProperties, 0));

    size_t N = (freeMem * 0.9 / 2) / sizeof(double);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    if (blocksPerGrid > deviceProperties.maxGridSize[0])
    {
        blocksPerGrid = deviceProperties.maxGridSize[0];
    }

    printf("Running %d blocks with %d threads/block over %lu elements\n",
        blocksPerGrid, threadsPerBlock, N);

    floatSample = (float *)malloc(sizeof(float) * sampleLength);
    doubleSample = (double *)malloc(sizeof(double) * sampleLength);

    // Define run_test as a templated lambda with mean calculations included
    auto run_test = [=]<typename T>(T* sample, double& meanToDeviceTime, 
                                   double& meanKernelTime, double& meanFromDeviceTime)
    {
        meanToDeviceTime = meanKernelTime = meanFromDeviceTime = 0.0;

        for (int run = 0; run < nRuns; run++)
        {
            T *h_inputs, *h_outputs;
            T *d_inputs, *d_outputs;
            long toDeviceTime, kernelTime, fromDeviceTime;
            clock_t ops_start, ops_end;

            h_inputs = (T*)malloc(sizeof(T) * N);
            h_outputs = (T*)malloc(sizeof(T) * N);
            gpuErrchk(hipMalloc((void**)&d_inputs, sizeof(T) * N));
            gpuErrchk(hipMalloc((void**)&d_outputs, sizeof(T) * N));

            for (int i = 0; i < N; i++)
            {
                h_inputs[i] = (T)i;
            }

            ops_start = clock();
            gpuErrchk(hipMemcpy(d_inputs, h_inputs, sizeof(T) * N, hipMemcpyHostToDevice));
            ops_end = clock();
            toDeviceTime = ops_end - ops_start;

            ops_start = clock();
            lots_of_compute<T><<<blocksPerGrid, threadsPerBlock>>>(d_inputs, N, nKernelIters, d_outputs);
            gpuErrchk(hipDeviceSynchronize());
            ops_end = clock();
            kernelTime = ops_end - ops_start;

            ops_start = clock();
            gpuErrchk(hipMemcpy(h_outputs, d_outputs, sizeof(T) * N, hipMemcpyDeviceToHost));
            ops_end = clock();
            fromDeviceTime = ops_end - ops_start;

            for (int i = 0; i < sampleLength; i++)
            {
                sample[i] = h_outputs[i];
            }

            meanToDeviceTime += toDeviceTime;
            meanKernelTime += kernelTime;
            meanFromDeviceTime += fromDeviceTime;

            gpuErrchk(hipFree(d_inputs));
            gpuErrchk(hipFree(d_outputs));
            free(h_inputs);
            free(h_outputs);
        }

        // Calculate means inside the lambda
        meanToDeviceTime /= (nRuns * CLOCKS_PER_SEC);
        meanKernelTime /= (nRuns * CLOCKS_PER_SEC);
        meanFromDeviceTime /= (nRuns * CLOCKS_PER_SEC);
    };

    // Run benchmarks using the lambda
    run_test.template operator()<float>(floatSample, 
        meanFloatToDeviceTime, meanFloatKernelTime, meanFloatFromDeviceTime);
    
    run_test.template operator()<double>(doubleSample,
        meanDoubleToDeviceTime, meanDoubleKernelTime, meanDoubleFromDeviceTime);

    printf("For single-precision floating point, mean times for:\n");
    printf("  Copy to device:   %f s\n", meanFloatToDeviceTime);
    printf("  Kernel execution: %f s\n", meanFloatKernelTime);
    printf("  Copy from device: %f s\n", meanFloatFromDeviceTime);
    printf("For double-precision floating point, mean times for:\n");
    printf("  Copy to device:   %f s (%.2fx slower than single-precision)\n",
        meanDoubleToDeviceTime,
        meanDoubleToDeviceTime / meanFloatToDeviceTime);
    printf("  Kernel execution: %f s (%.2fx slower than single-precision)\n",
        meanDoubleKernelTime,
        meanDoubleKernelTime / meanFloatKernelTime);
    printf("  Copy from device: %f s (%.2fx slower than single-precision)\n",
        meanDoubleFromDeviceTime,
        meanDoubleFromDeviceTime / meanFloatFromDeviceTime);

    return 0;
}
