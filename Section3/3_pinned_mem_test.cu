#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <chrono>

// Function to measure memory transfer time
float measureTransferTime(float* src, float* dst, size_t nbytes, hipMemcpyKind kind) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	hipMemcpy(dst, src, nbytes, kind);
	hipEventRecord(stop);
	hipEventSynchronize(stop);  // We need this to ensure events are ready to be read
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return milliseconds;
}

int main(int argc, char **argv) 
{   								
	// memory size   128 MBs
	int isize = 1<<25;   
	int nbytes = isize * sizeof(float);
											
	printf("Memory size: %d MB\n", nbytes / (1024 * 1024));
											
	// Warm-up run to avoid initialization overhead
	{
		float *temp_pinned;
		float *temp_device;
		hipHostMalloc((float **)&temp_pinned, nbytes, hipHostMallocDefault);
		hipMalloc((float **)&temp_device, nbytes);
		hipMemcpy(temp_device, temp_pinned, nbytes, hipMemcpyHostToDevice);
		hipHostFree(temp_pinned);
		hipFree(temp_device);
	}
											
	// Allocate pinned memory
	float *h_pinned;
	hipHostMalloc((float **)&h_pinned, nbytes, hipHostMallocDefault);
											
	// Allocate regular (unpinned) memory
	float *h_unpinned = (float *)malloc(nbytes);
											
	// Allocate device memory   
	float *d_a; 
	hipMalloc((float **)&d_a, nbytes);
									
	// Initialize both host memories
	for(int i=0; i<isize; i++) {
		h_pinned[i] = 7;
		h_unpinned[i] = 7;
	}
									
	// Measure pinned memory transfer
	printf("\nPinned Memory Transfer:\n");
	float pinned_to_device = measureTransferTime(h_pinned, d_a, nbytes, hipMemcpyHostToDevice);
	float pinned_from_device = measureTransferTime(d_a, h_pinned, nbytes, hipMemcpyDeviceToHost);
	printf("Host to Device: %.3f ms\n", pinned_to_device);
	printf("Device to Host: %.3f ms\n", pinned_from_device);
	printf("Total Transfer Time: %.3f ms\n", pinned_to_device + pinned_from_device);
									
	// Measure unpinned memory transfer
	printf("\nUnpinned Memory Transfer:\n");
	float unpinned_to_device = measureTransferTime(h_unpinned, d_a, nbytes, hipMemcpyHostToDevice);
	float unpinned_from_device = measureTransferTime(d_a, h_unpinned, nbytes, hipMemcpyDeviceToHost);
	printf("Host to Device: %.3f ms\n", unpinned_to_device);
	printf("Device to Host: %.3f ms\n", unpinned_from_device);
	printf("Total Transfer Time: %.3f ms\n", unpinned_to_device + unpinned_from_device);
									
	// Calculate speedup
	float total_pinned = pinned_to_device + pinned_from_device;
	float total_unpinned = unpinned_to_device + unpinned_from_device;
	float speedup = total_unpinned / total_pinned;
	printf("\nPinned memory is %.2fx faster than unpinned memory\n", speedup);
									
	// Free memory   
	hipFree(d_a);
	hipHostFree(h_pinned);
	free(h_unpinned);
									
	// Reset device    
	hipDeviceReset();   
	return EXIT_SUCCESS;
}